#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <device_atomic_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <vector_functions.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <stdio.h>
#include <time.h>

#include <stdio.h>
#include <stdlib.h>
#include <chrono>


#include "modelBER_params.cuh"


__constant__ unsigned long long int kRandomSeed;		/// <summary>�����̃V�[�h�l</summary>



#define CUDA_SAFE_CALL(func) \
{ \
     hipError_t err = (func); \
     if (err != hipSuccess) \
	 { \
         fprintf(stderr, "[Error] %s (error code: %d) at %s line %d\n", hipGetErrorString(err), err, __FILE__, __LINE__); \
         exit(err); \
	 } \
}


#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else

__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}


#endif

__device__ inline float calcCurieFromCu(float cu)
{
	return (2 * J_FE_FE * 4 * (1 - cu) * (G_FE - 1)  * (G_FE - 1) * S_FE * (S_FE + 1)) / (3 * K_B);
}

__device__ inline float calcCuFromCurie(float temp_curie)
{
	return -temp_curie * (3 * K_B) / (2 * J_FE_FE * 4 * (G_FE - 1) *(G_FE - 1) * S_FE * (S_FE + 1)) + 1;
}

__host__ __device__ inline float convertTempFromAP(int ap_count)
{
	//return fmax(TEMP_AMBIENT, TEMP_CURIE_MEAN - THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9 * ap_count);
	float temp = TEMP_CURIE_MEAN - THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9F * ap_count;
	return temp < TEMP_AMBIENT ? TEMP_AMBIENT : temp;
}


__device__ inline float calcBrillouin(float s, float x)
{

	float o2 = 1 / (2 * S_FE);
	float o1 = (2 * S_FE + 1) * o2;
	return o1 / tanh(o1 * x) - o2 / tanh(o2 * x);
	
}

__device__ float calc_sFe_Mean(float temp, float cu)
{
	float vl = S_FE_MEAN_ERROR;
	float vr = S_FE_MEAN_MAX;
	float v;
	float dxdv = (S_FE * 2 * J_FE_FE * 4 * (1 - cu) * (G_FE - 1)*(G_FE - 1)) / (K_B * temp);
	do
	{
		v = (vl + vr) / 2.0;
		float br = S_FE *(2 * S_FE + 1) / (2 * S_FE) / tanh((2 * S_FE + 1) / (2 * S_FE) * dxdv * v) - S_FE  / (2 * S_FE) / tanh(1 / (2 * S_FE) * dxdv * v);
		float f = v - br;
		//float f = v - S_FE * calcBrillouin(S_FE, dxdv * v);
		if (f < 0)
			vl = v;
		else
			vr = v;

	} while (fabs((vl - vr) / v) > S_FE_MEAN_ERROR);
	
	return v;
}

__device__ float calcKb(float temp, float hw, float cu)
{

	float temp_curie = calcCurieFromCu(cu);
	if (temp >= temp_curie) return 0;

	float dFeFe = BULK_D_FE_FE * KU_KBULK;

	float s = calc_sFe_Mean(temp, cu);

	float total_atom_number = 1 / ((FE  * (1 - cu) * V_FE) + ((1 - FE)*(1 - cu) * V_PT) + cu * V_CU);
	float ku = total_atom_number * FE * (1 - cu)* (4 * (1 - cu))  * dFeFe  * s * s;
	float hc = 2 * ((4 * (1 - cu))  * dFeFe  * s) / (M_B * G_FE);
	if (hw < 0 && hc <= fabs(hw)) return 0;

	float kb = (ku * GRAIN_VOLUME) / K_B / temp * (1 + hw / hc) * (1 + hw / hc);
	return kb;
}

__device__ void calcKb(float temp, float hw, float cu, float tc, float &kbp, float &kbm)
{
	kbm = kbp = 0;
	if (tc <= temp) return;

	const float dFeFe = BULK_D_FE_FE * KU_KBULK;

	float s = calc_sFe_Mean(temp, cu);
	float total_atom_number = 1 / ((FE  * (1 - cu) * V_FE) + ((1 - FE)*(1 - cu) * V_PT) + cu * V_CU);
	float ku = total_atom_number * FE * (1 - cu)* (4 * (1 - cu))  * dFeFe  * s * s;
	float hc = 2 * ((4 * (1 - cu))  * dFeFe  * s) / (M_B * G_FE);
	kbp = (ku * GRAIN_VOLUME) / K_B / temp * (1 + hw / hc) * (1 + hw / hc);
	kbm = hc <= hw ? 0 : (ku * GRAIN_VOLUME) / K_B / temp * (1 - hw / hc) * (1 - hw / hc);
}

__global__
void calcKbListKernel(float *kb_minus_list, float *kb_plus_list, int kb_list_size, int offset, float hw)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= kb_list_size) return;

	float cu = calcCuFromCurie(TEMP_CURIE_MEAN);
	float temp = convertTempFromAP(i - offset);
	float kbp, kbm;
	calcKb(temp, hw, cu, TEMP_CURIE_MEAN, kbp, kbm);
	kb_plus_list[i] = kbp;
	kb_minus_list[i] = kbm;
}

__host__
void calcKbListHost(FILE *fp, float hw)
{
	int bit_ap = (int)(BIT_PITCH * 1e-9 / TAU_AP / LINER_VELOCITY);
	int kb_list_count = bit_ap * 4;
	int offset = bit_ap;

	int thread_count = (int)(fmax(sqrt(kb_list_count),THREAD_NUM));
	int block_count = (kb_list_count / thread_count + 1);

	CUDA_SAFE_CALL(hipSetDevice(CUDA_DEVICE_NUM));

	thrust::host_vector<float> host_kb_m_list(kb_list_count);
	thrust::host_vector<float> host_kb_p_list(kb_list_count);
	thrust::device_vector<float> dev_kb_m_list(kb_list_count);
	thrust::device_vector<float> dev_kb_p_list(kb_list_count);

	calcKbListKernel << <thread_count, block_count >> >(
		thrust::raw_pointer_cast(dev_kb_m_list.data()),
		thrust::raw_pointer_cast(dev_kb_p_list.data()),
		kb_list_count,
		offset,
		hw);

	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	host_kb_m_list = dev_kb_m_list;
	host_kb_p_list = dev_kb_p_list;


	fprintf(fp , "%s\t%s\t%s\t%s\t%s\t%s\t%s\t%s\n",
		"AP Count",
		"Temp(K)",
		"Time(ns)",
		"Distance(nm)",
		"Kb+",
		"Kb-",
		"p+",
		"p-");
	for (int i = 0; i < kb_list_count; i++)
	{
		int ap = i - offset;
		fprintf(fp, "%d\t%e\t%e\t%e\t%e\t%e\t%e\t%e\n",
			ap,
			convertTempFromAP(i - offset),
			ap * TAU_AP * 1e+9,
			ap * TAU_AP * 1e+9 * LINER_VELOCITY,
			host_kb_m_list[i],
			host_kb_p_list[i],
			exp(-host_kb_m_list[i]),
			exp(-host_kb_p_list[i]));

	}

}




#if BER_ALGORITHM == 0


// ###
// ###  �P�������e�J�����@
// ###
// #########################################################################################################################


__global__ void calcContinusBitErrorRateKernel(int *ber_list, int ber_list_count, float hw)
{
	int thread_number = threadIdx.x + blockIdx.x * blockDim.x;
	float grain_tc[GRAIN_COUNT];			// �O���C�����Ƃ�Tc
	float grain_cu[GRAIN_COUNT];			// �O���C�����Ƃ�Cu�g��
	float grain_area[GRAIN_COUNT];			// �O���C�����Ƃ̖ʐ�
	//float grain_ku_kum[GRAIN_COUNT];			// �O���C�����Ƃ�Ku/Kum
	int grain_dir[GRAIN_COUNT];				// �O���C���̎����̌��� (1 = �t�����A-1 = ������)
	float grain_size_mu = log((GRAIN_MEAN * GRAIN_MEAN) / sqrt(GRAIN_SD * GRAIN_SD + GRAIN_MEAN * GRAIN_MEAN));  					  // �O���C���T�C�Y���U�̃�
	float grain_size_sigma = (sqrt(log((GRAIN_SD * GRAIN_SD) / (GRAIN_MEAN * GRAIN_MEAN) + 1)));									  // �O���C���T�C�Y���U�̃�
	const int hw_switch_ap = (int)(BIT_PITCH / LINER_VELOCITY *1.0e-9 * F0_AP);														  // �������E�������I���^�C�~���O
	const int attempt_offset = (int)(TEMP_CURIE_MEAN * TEMP_CURIE_SD * 2 / (THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9));	  // Tcm�ȑO�̗\���V�~�����[�V����


	hiprandStateMRG32k3a rand_stat;			// �����X�e�[�^�X	
	hiprand_init(kRandomSeed, thread_number, ber_list_count * GRAIN_COUNT, &rand_stat);


	for (int i = 0; i < GRAIN_COUNT; i++)
	{
		grain_tc[i] = hiprand_normal_double(&rand_stat) * TEMP_CURIE_SD * TEMP_CURIE_MEAN + TEMP_CURIE_MEAN;
		grain_cu[i] = calcCuFromCurie(grain_tc[i]);
		float a = hiprand_log_normal_double(&rand_stat, grain_size_mu, grain_size_sigma);
		grain_area[i] = a * a;
		//grain_ku_kum[i] = 1;
		grain_dir[i] = 1;
	}

	float signed_hw = hw;
	for (int i = -attempt_offset; i < ber_list_count; i++)
	{
		float signal_power = 0;

		float temp = convertTempFromAP(i);

		if (i == 0 || i == hw_switch_ap)
			signed_hw = -signed_hw;


		for (int k = 0; k < GRAIN_COUNT; k++)
		{
			// �������]����m��
			// hw = -1 = ������
			// grain_dir = 1 �t����
			// hw * grain_dir = -1 hw�����ւ̔��]�m��
			if (temp > grain_tc[k]) continue;

			float rev_prob = exp(-calcKb(temp, signed_hw * grain_dir[k], grain_cu[k]) * grain_area[k]);
			float dice = hiprand_uniform(&rand_stat);
			if (rev_prob > dice)
				grain_dir[k] = -grain_dir[k];
			if (grain_dir[k] < 0)
				signal_power += grain_area[k];
		}

		if (READABLE_THRETH > signal_power && 0 <= i && i < ber_list_count)
			atomicAdd(&ber_list[i], 1);
	}
}

void calcContinusBitErrorRateHost(float *bER_list, int bER_list_count, float hw)
{
	int *dev_be_list;
	int *be_list = (int*)malloc(sizeof(int) * bER_list_count);
	unsigned long long int random_seed = (unsigned long long int)(time(NULL));

	for (int i = 0; i < bER_list_count; i++)
		be_list[i] = 0;

	CUDA_SAFE_CALL(hipSetDevice(CUDA_DEVICE_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_be_list, sizeof(int) * bER_list_count));
	CUDA_SAFE_CALL(hipMemcpy(dev_be_list, be_list, sizeof(int) * bER_list_count, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(kRandomSeed), &random_seed, sizeof(unsigned long long int), hipMemcpyHostToDevice));


	calcContinusBitErrorRateKernel << <CUDA_BLOCK_COUNT, CUDA_THREAD_COUNT >> >(dev_be_list, bER_list_count, hw);
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(hipMemcpy(be_list, dev_be_list, sizeof(int) * bER_list_count, hipMemcpyDeviceToHost));

	for (int i = 0; i < bER_list_count; i++)
	{
		bER_list[i] = (float)be_list[i] / BIT_COUNT;
	}
	CUDA_SAFE_CALL(hipFree(dev_be_list));
	free(be_list);
}

__global__ void calcMidLastBitErrorRateKernel(int *mid_be_list, int *last_be_list, float hw)
{
	int thread_number = threadIdx.x + blockIdx.x * blockDim.x;
	float grain_tc[GRAIN_COUNT];			// �O���C�����Ƃ�Tc
	float grain_cu[GRAIN_COUNT];			// �O���C�����Ƃ�Cu�g��
	float grain_area[GRAIN_COUNT];			// �O���C�����Ƃ̖ʐ�
	//float grain_ku_kum[GRAIN_COUNT];			// �O���C�����Ƃ�Ku/Kum
	int grain_dir[GRAIN_COUNT];				// �O���C���̎����̌��� (1 = �t�����A-1 = ������)
	float grain_size_mu = log((GRAIN_MEAN * GRAIN_MEAN) / sqrt(GRAIN_SD * GRAIN_SD + GRAIN_MEAN * GRAIN_MEAN));  					  // �O���C���T�C�Y���U�̃�
	float grain_size_sigma = (sqrt(log((GRAIN_SD * GRAIN_SD) / (GRAIN_MEAN * GRAIN_MEAN) + 1)));									  // �O���C���T�C�Y���U�̃�
	const int hw_switch_ap = (int)(BIT_PITCH / LINER_VELOCITY *1.0e-9 * F0_AP);														  // �������E�������I���^�C�~���O
	const int attempt_offset = (int)(TEMP_CURIE_MEAN * TEMP_CURIE_SD * 2 / (THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9));	  // Tcm�ȑO�̗\���V�~�����[�V����
	const int last_attempt = hw_switch_ap * 2 + attempt_offset;



	hiprandStateMRG32k3a rand_stat;			// �����X�e�[�^�X	
	hiprand_init(kRandomSeed, thread_number, last_attempt * GRAIN_COUNT, &rand_stat);


	for (int i = 0; i < GRAIN_COUNT; i++)
	{
		grain_tc[i] = hiprand_normal_double(&rand_stat) * TEMP_CURIE_SD * TEMP_CURIE_MEAN + TEMP_CURIE_MEAN;
		grain_cu[i] = calcCuFromCurie(grain_tc[i]);
		float a = hiprand_log_normal_double(&rand_stat, grain_size_mu, grain_size_sigma);
		grain_area[i] = a * a;
		//grain_ku_kum[i] = 1;
		grain_dir[i] = 1;
	}
	mid_be_list[thread_number] = 0;
	last_be_list[thread_number] = 0;

	float signed_hw = hw;
	for (int i = -attempt_offset; i < last_attempt; i++)
	{
		float signal_power = 0;

		float temp = convertTempFromAP(i);

		if (i == 0 || i == hw_switch_ap)
			signed_hw = -signed_hw;

		for (int k = 0; k < GRAIN_COUNT; k++)
		{
			// �������]����m��
			// hw = -1 = ������
			// grain_dir = 1 �t����
			// hw * grain_dir = -1 hw�����ւ̔��]�m��
			if (temp > grain_tc[k]) continue;

			float rev_prob = exp(-calcKb(temp, signed_hw * grain_dir[k], grain_cu[k]) * grain_area[k]);
			float dice = hiprand_uniform(&rand_stat);
			if (rev_prob > dice)
				grain_dir[k] = -grain_dir[k];
			if (grain_dir[k] < 0)
				signal_power += grain_area[k];
		}

		if (i == hw_switch_ap - 1 && READABLE_THRETH > signal_power)
			mid_be_list[thread_number] = 1;
		if (i == last_attempt - 1 && READABLE_THRETH > signal_power)
			last_be_list[thread_number] = 1;
	}

}

void calcMidLastBitErrorRateHost(float *mid_bER, float *last_bER, float hw)
{
	const int list_size = BIT_COUNT;
	int *mid_be_list = (int*)malloc(sizeof(int) * list_size);
	int *last_be_list = (int*)malloc(sizeof(int) * list_size);
	int *dev_mid_be_list = NULL;
	int *dev_last_be_list = NULL;

	CUDA_SAFE_CALL(hipSetDevice(CUDA_DEVICE_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_mid_be_list, sizeof(int) * list_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_last_be_list, sizeof(int) * list_size));

	CUDA_SAFE_CALL(hipMemcpy(dev_mid_be_list, mid_be_list, sizeof(int) * list_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_last_be_list, last_be_list, sizeof(int) * list_size, hipMemcpyHostToDevice));

	calcMidLastBitErrorRateKernel << <CUDA_BLOCK_COUNT, CUDA_THREAD_COUNT >> >(dev_mid_be_list, dev_last_be_list, hw);
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(hipMemcpy(mid_be_list, dev_mid_be_list, sizeof(int) * list_size, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(last_be_list, dev_last_be_list, sizeof(int) * list_size, hipMemcpyDeviceToHost));

	float temp_mid_bER = 0;
	float temp_last_bER = 0;
	for (int i = 0; i < list_size; i++)
	{
		temp_mid_bER += mid_be_list[i];
		temp_last_bER += last_be_list[i];
	}
	temp_mid_bER /= list_size;
	temp_last_bER /= list_size;

	*mid_bER = temp_mid_bER;
	*last_bER = temp_last_bER;

	CUDA_SAFE_CALL(hipFree(dev_mid_be_list));
	CUDA_SAFE_CALL(hipFree(dev_last_be_list));
	free(mid_be_list);
	free(last_be_list);

}



#else

// ###
// ###  �m���@�E�@�p�^�[���@
// ###
// #########################################################################################################################

__device__ inline float calcPattern(float *grain_area, float *grain_prob)
{
	float bit_error_rate = 0;
#define GA_(n) (0)
#define GAM(n) (grain_area[n])
#define GP_(n) (grain_prob[n])
#define GPM(n) (1 - grain_prob[n])

#if GRAIN_COUNT == 1
	if (grain_area[0] < READABLE_THRETH) bit_error_rate = grain_prob[0];

#elif GRAIN_COUNT == 4
	if (GA_(0) + GA_(1) + GA_(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GP_(1) * GP_(2) * GP_(3);  //  0
	if (GAM(0) + GA_(1) + GA_(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GP_(1) * GP_(2) * GP_(3);  //  1
	if (GA_(0) + GAM(1) + GA_(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GPM(1) * GP_(2) * GP_(3);  //  2
	if (GAM(0) + GAM(1) + GA_(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GPM(1) * GP_(2) * GP_(3);  //  3
	if (GA_(0) + GA_(1) + GAM(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GP_(1) * GPM(2) * GP_(3);  //  4
	if (GAM(0) + GA_(1) + GAM(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GP_(1) * GPM(2) * GP_(3);  //  5
	if (GA_(0) + GAM(1) + GAM(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GPM(1) * GPM(2) * GP_(3);  //  6
	if (GAM(0) + GAM(1) + GAM(2) + GA_(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GPM(1) * GPM(2) * GP_(3);  //  7
	if (GA_(0) + GA_(1) + GA_(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GP_(1) * GP_(2) * GPM(3);  //  8
	if (GAM(0) + GA_(1) + GA_(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GP_(1) * GP_(2) * GPM(3);  //  9
	if (GA_(0) + GAM(1) + GA_(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GPM(1) * GP_(2) * GPM(3);  // 10
	if (GAM(0) + GAM(1) + GA_(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GPM(1) * GP_(2) * GPM(3);  // 11
	if (GA_(0) + GA_(1) + GAM(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GP_(1) * GPM(2) * GPM(3);  // 12
	if (GAM(0) + GA_(1) + GAM(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GP_(1) * GPM(2) * GPM(3);  // 13
	if (GA_(0) + GAM(1) + GAM(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GP_(0) * GPM(1) * GPM(2) * GPM(3);  // 14
	if (GAM(0) + GAM(1) + GAM(2) + GAM(3) < READABLE_THRETH) bit_error_rate += GPM(0) * GPM(1) * GPM(2) * GPM(3);  // 15
#else
#error Not implement for this GRAIN_COUNT pattern


#endif
	
#undef GA_
#undef GAM
#undef GP_
#undef GPM
	

	return bit_error_rate;
}

__global__ void calcContinusBitErrorRateKernel(float *ber_list, int ber_list_count, float hw)
{

	int thread_number = threadIdx.x + blockIdx.x * blockDim.x;
	float grain_prob[GRAIN_COUNT];				// �O���C���̋t�����Ɍ����Ă���m��
	float grain_tc[GRAIN_COUNT];				// �O���C�����Ƃ�Tc
	float grain_cu[GRAIN_COUNT];				// �O���C�����Ƃ�Cu�g��
	float grain_area[GRAIN_COUNT];				// �O���C�����Ƃ̖ʐ�
	//float grain_ku_kum[GRAIN_COUNT];			// �O���C�����Ƃ�Ku/Kum
	float grain_size_mu = log((GRAIN_MEAN * GRAIN_MEAN) / sqrt(GRAIN_SD * GRAIN_SD + GRAIN_MEAN * GRAIN_MEAN));  					  // �O���C���T�C�Y���U�̃�
	float grain_size_sigma = (sqrt(log((GRAIN_SD * GRAIN_SD) / (GRAIN_MEAN * GRAIN_MEAN) + 1)));									  // �O���C���T�C�Y���U�̃�
	const int hw_switch_ap = (int)(BIT_PITCH / LINER_VELOCITY *1.0e-9 * F0_AP);														  // �������E�������I���^�C�~���O
	const int attempt_offset = (int)(TEMP_CURIE_MEAN * TEMP_CURIE_SD * 2 / (THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9));	  // Tcm�ȑO�̗\���V�~�����[�V����


	hiprandStateMRG32k3a rand_stat;			// �����X�e�[�^�X	
	hiprand_init(kRandomSeed, thread_number, ber_list_count * GRAIN_COUNT, &rand_stat);


	for (int i = 0; i < GRAIN_COUNT; i++)
	{
		grain_tc[i] = hiprand_normal (&rand_stat) * TEMP_CURIE_SD * TEMP_CURIE_MEAN + TEMP_CURIE_MEAN;
		grain_cu[i] = calcCuFromCurie(grain_tc[i]);
		float a = hiprand_log_normal(&rand_stat, grain_size_mu, grain_size_sigma);
		grain_area[i] = a * a;
		//grain_ku_kum[i] = 1;
		grain_prob[i] = INITIAL_MAG_PROB;
	}


	for (int i = -attempt_offset; i < ber_list_count; i++)
	{

		float temp = TEMP_CURIE_MEAN - THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9 * i;
		if (temp < TEMP_AMBIENT)
			temp = TEMP_AMBIENT;



		for (int k = 0; k < GRAIN_COUNT; k++)
		{

			if (temp > grain_tc[k]) continue;

			float kbm, kbp;
			calcKb(temp, hw, grain_cu[k],grain_tc[k], kbp, kbm);

			float prob_neg = 0 <= i && i < hw_switch_ap ? exp(-kbp * grain_area[k]) : exp(-kbm * grain_area[k]);
			float prob_pog = 0 <= i && i < hw_switch_ap ? exp(-kbm * grain_area[k]) : exp(-kbp * grain_area[k]);
			grain_prob[k] = prob_neg * (1 - grain_prob[k]) + (1 - prob_pog) * grain_prob[k];
		}



		float ber = calcPattern(grain_area, grain_prob);
		atomicAdd(&ber_list[i], ber);
	}
}

void calcContinusBitErrorRateHost(float *bER_list, int bER_list_count, float hw)
{
	float *dev_ber_list;
	unsigned long long int random_seed = (unsigned long long int)(time(NULL));

	for (int i = 0; i < bER_list_count; i++)
		bER_list[i] = 0;

	CUDA_SAFE_CALL(hipSetDevice(CUDA_DEVICE_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_ber_list, sizeof(int) * bER_list_count));
	CUDA_SAFE_CALL(hipMemcpy(dev_ber_list, bER_list, sizeof(int) * bER_list_count, hipMemcpyHostToDevice));

	//CUDA_SAFE_CALL(hipMemcpy(&kRandomSeed, &random_seed, sizeof(unsigned long long int), hipMemcpyHostToDevice));
	
	calcContinusBitErrorRateKernel <<< CUDA_BLOCK_COUNT, CUDA_THREAD_COUNT >>>(dev_ber_list, bER_list_count, hw);

	CUDA_SAFE_CALL(hipGetLastError());


	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(bER_list, dev_ber_list, sizeof(int) * bER_list_count, hipMemcpyDeviceToHost));

	for (int i = 0; i < bER_list_count; i++)
	{
		bER_list[i] = bER_list[i] / BIT_COUNT;
	}
	CUDA_SAFE_CALL(hipFree(dev_ber_list));
}

__global__ void calcMidLastBitErrorRateKernel(float *mid_be_list, float *last_be_list, float hw)
{

	int thread_number = threadIdx.x + blockIdx.x * blockDim.x;
	float grain_prob[GRAIN_COUNT];			// �O���C���̋t�����Ɍ����Ă���m��
	float grain_tc[GRAIN_COUNT];			// �O���C�����Ƃ�Tc
	float grain_cu[GRAIN_COUNT];			// �O���C�����Ƃ�Cu�g��
	float grain_area[GRAIN_COUNT];			// �O���C�����Ƃ̖ʐ�
	//float grain_ku_kum[GRAIN_COUNT];			// �O���C�����Ƃ�Ku/Kum
	float grain_size_mu = log((GRAIN_MEAN * GRAIN_MEAN) / sqrt(GRAIN_SD * GRAIN_SD + GRAIN_MEAN * GRAIN_MEAN));  					  // �O���C���T�C�Y���U�̃�
	float grain_size_sigma = (sqrt(log((GRAIN_SD * GRAIN_SD) / (GRAIN_MEAN * GRAIN_MEAN) + 1)));									  // �O���C���T�C�Y���U�̃�
	const int hw_switch_ap = (int)(BIT_PITCH / LINER_VELOCITY *1.0e-9 * F0_AP);														  // �������E�������I���^�C�~���O
	const int attempt_offset = (int)(TEMP_CURIE_MEAN * TEMP_CURIE_SD * 2 / (THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9));	  // Tcm�ȑO�̗\���V�~�����[�V����
	const int last_attempt = hw_switch_ap * 2 + attempt_offset;


	hiprandStateMRG32k3a rand_stat;			// �����X�e�[�^�X	
	hiprand_init(kRandomSeed, thread_number, last_attempt * GRAIN_COUNT, &rand_stat);

	for (int i = 0; i < GRAIN_COUNT; i++)
	{
		grain_tc[i] = hiprand_normal(&rand_stat) * TEMP_CURIE_SD * TEMP_CURIE_MEAN + TEMP_CURIE_MEAN;
		grain_cu[i] = calcCuFromCurie(grain_tc[i]);
		float a = hiprand_log_normal(&rand_stat, grain_size_mu, grain_size_sigma);
		grain_area[i] = a * a;
		grain_prob[i] = INITIAL_MAG_PROB;
		//grain_ku_kum[i] = 1;
	}
	mid_be_list[thread_number] = 0;
	last_be_list[thread_number] = 0;

	for (int i = -attempt_offset; i < last_attempt; i++)
	{

		float temp = TEMP_CURIE_MEAN - THERMAL_GRADIENT * LINER_VELOCITY * TAU_AP * 1.0e+9F * i;
		if (temp < TEMP_AMBIENT)
			temp = TEMP_AMBIENT;

		for (int k = 0; k < GRAIN_COUNT; k++)
		{
			if (temp > grain_tc[k]) continue;

			float kbm, kbp;
			calcKb(temp, hw, grain_cu[k],grain_tc[k], kbp, kbm);
			//kbp = calcKb(temp, hw, grain_cu[k]);
			//kbm = calcKb(temp, -hw, grain_cu[k]);

			float prob_neg = 0 <= i && i < hw_switch_ap ? exp(-kbp * grain_area[k]) : exp(-kbm * grain_area[k]);
			float prob_pog = 0 <= i && i < hw_switch_ap ? exp(-kbm * grain_area[k]) : exp(-kbp * grain_area[k]);
			grain_prob[k] = prob_neg * (1 - grain_prob[k]) + (1 - prob_pog) * grain_prob[k];
		}

		if (i == hw_switch_ap - 1) // EAW�p�ɒ�����
			mid_be_list[thread_number] = calcPattern(grain_area, grain_prob);
			
	}

	last_be_list[thread_number] = calcPattern(grain_area, grain_prob);

}

void calcMidLastBitErrorRateHost(float *mid_bER, float *last_bER, float hw)
{
	const int list_size = BIT_COUNT;
	float *mid_be_list = (float*)malloc(sizeof(float) * list_size);
	float *last_be_list = (float*)malloc(sizeof(float) * list_size);
	float *dev_mid_be_list = NULL;
	float *dev_last_be_list = NULL;
	unsigned long long int random_seed = (unsigned long long int)(time(NULL));

	CUDA_SAFE_CALL(hipSetDevice(CUDA_DEVICE_NUM));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_mid_be_list, sizeof(float) * list_size));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_last_be_list, sizeof(float) * list_size));

	CUDA_SAFE_CALL(hipMemcpy(dev_mid_be_list, mid_be_list, sizeof(float) * list_size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_last_be_list, last_be_list, sizeof(float) * list_size, hipMemcpyHostToDevice));
	//CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&kRandomSeed), &random_seed, sizeof(unsigned long long int)));


	calcMidLastBitErrorRateKernel <<<CUDA_BLOCK_COUNT, CUDA_THREAD_COUNT>>>(dev_mid_be_list, dev_last_be_list, hw);
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(mid_be_list, dev_mid_be_list, sizeof(float) * list_size, hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(last_be_list, dev_last_be_list, sizeof(float) * list_size, hipMemcpyDeviceToHost));

	
	float temp_mid_bER = 0;
	float temp_last_bER = 0;

	/*// �j�R�C�`���_�N�V�����A�����x�H
	float half = list_size / 2;
	for (int i = 0; i < log2(list_size)-1; i++)
	{
		for (int k = 0; k < half-1; k++)
		{
			mid_be_list[k] = (mid_be_list[2 * k] + mid_be_list[2 * k + 1]) / 2.0;
			last_be_list[k] = (last_be_list[2 * k] + last_be_list[2 * k + 1]) / 2.0;
		}
		half /= 2.0;
	}

	temp_mid_bER = mid_be_list[0];
	temp_last_bER = last_be_list[0];
	*/


	
	// �P�����_�N�V����
	for (int i = 0; i < list_size; i++)
	{
		temp_mid_bER += mid_be_list[i];
		temp_last_bER += last_be_list[i];
	}

	temp_mid_bER /= list_size;
	temp_last_bER /= list_size;
	
	

	/*
	// thrust GPU���_�N�V����
	thrust::device_ptr<float> dev_mid_be_ptr(dev_mid_be_list);
	thrust::device_ptr<float> dev_last_be_ptr(dev_last_be_list);

	temp_mid_bER = thrust::reduce(dev_mid_be_ptr, dev_mid_be_ptr + list_size);
	temp_last_bER = thrust::reduce(dev_last_be_ptr, dev_last_be_ptr + list_size);
	temp_mid_bER /= list_size;
	temp_last_bER /= list_size;
	*/

	*mid_bER = temp_mid_bER;
	*last_bER = temp_last_bER;

	CUDA_SAFE_CALL(hipFree(dev_mid_be_list));
	CUDA_SAFE_CALL(hipFree(dev_last_be_list));
	free(mid_be_list);
	free(last_be_list);

}

#endif



void makeHwBerList(FILE *fp)
{
	fprintf(fp, "Hw[kOe]\tbER\tbER(WE)\tbER(EAW)\n");
	for (int i = 0; i < HW_LIST_SIZE; i++)
	{
		printf("%d / %d \r", i, HW_LIST_SIZE);

		float hw = (HW_LAST- HW_FIRST) * i / HW_LIST_SIZE + HW_FIRST;
		float mid_bER = 0;
		float last_bER = 0;
		calcMidLastBitErrorRateHost(&mid_bER, &last_bER, hw);

		fprintf(fp, "%f\t%.10e\t%.10e\t%.10e\n", hw * 1e-3, last_bER, mid_bER, last_bER - mid_bER);
	}
}

void makeContinusBerList(FILE *fp)
{
	const int hw_switch_ap = (int)(BIT_PITCH / LINER_VELOCITY *1.0e-9 * F0_AP);														  // �������E�������I���^�C�~���O
	const int attempt_count = hw_switch_ap * 3;
	float ber[attempt_count];

	calcContinusBitErrorRateHost(ber, attempt_count, CBER_HW);

	fprintf(fp, "Count\tTemp\tTime(ns)\tbER\n");
	for (int i = 0; i < attempt_count; i++)
	{
		fprintf(fp, "%d\t%f\t%f\t%e\n", i,
			convertTempFromAP(i),
			i * TAU_AP * 1e+9,
			ber[i]);
	}
}


/*************************************
*
*/


void subKbList()
{
	FILE *fp = fopen("kb_list.txt", "w");
	calcKbListHost(fp, 10e+3);
	fclose(fp);
}

void subHwBER()
{
	
#if (BER_ALGORITHM == 1)

	FILE *fp = fopen("hw_list_prob.txt", "w");
#else
	FILE *fp = fopen("hw_list_pure.txt", "w");

#endif
	makeHwBerList(fp);
	fclose(fp);
}


void subContinusBER()
{
#if (BER_ALGORITHM == 1)

	FILE *fp = fopen("cber_list_prob.txt", "w");
#else
	FILE *fp = fopen("cber_list_pure.txt", "w");

#endif
	makeContinusBerList(fp);
	fclose(fp);
}


int main()
{
	auto start = std::chrono::system_clock::now();
	hipProfilerStart();




	auto end = std::chrono::system_clock::now();
	auto dur = end - start;
	auto msec = std::chrono::duration_cast<std::chrono::milliseconds>(dur).count();

	std::cout << "\n" << msec << " milli sec \n";
	hipProfilerStop();
    return 0;
}
